#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel function to add two arrays
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 1000000; // 1 million elements
    size_t size = n * sizeof(int);
    
    // Host arrays
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);
    int *h_c = (int*)malloc(size);
    
    // Initialize arrays
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    // Device arrays
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("Launching kernel with %d blocks x %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // Verify result
    int errors = 0;
    for (int i = 0; i < n; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            errors++;
            if (errors < 5) printf("Error at index %d: expected %d, got %d\n", i, h_a[i] + h_b[i], h_c[i]);
        }
    }
    
    if (errors == 0) {
        printf("Vector addition completed successfully! No errors found.\n");
    } else {
        printf("Found %d errors in the results\n", errors);
    }
    
    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}