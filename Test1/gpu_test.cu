#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    printf("Number of CUDA devices: %d\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total GPU memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    }
    
    return 0;
}

