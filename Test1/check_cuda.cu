#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    printf("CUDA library test:\n");
    
    // Test if we can access CUDA functions
    int runtime_version = 0;
    hipError_t error = hipRuntimeGetVersion(&runtime_version);
    
    if (error == hipSuccess) {
        printf("CUDA Runtime version: %d\n", runtime_version);
        
        int deviceCount;
        error = hipGetDeviceCount(&deviceCount);
        if (error == hipSuccess) {
            printf("Number of CUDA devices: %d\n", deviceCount);
        } else {
            printf("hipGetDeviceCount error: %s\n", hipGetErrorString(error));
        }
    } else {
        printf("hipRuntimeGetVersion error: %s\n", hipGetErrorString(error));
    }
    
    return 0;
}